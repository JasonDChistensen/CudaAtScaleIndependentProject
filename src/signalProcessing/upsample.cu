#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include "upsample.cuh"

namespace upSample {

void execute(float *d_Output, float const *d_Input, int numElements, int upsampleFactor);

float * allocateDeviceMemory(size_t numberOfSamples, size_t upsampleFactor)
{
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // Allocate the device output vector U
  float *d_U = NULL;
  err = hipMalloc((void **)&d_U, numberOfSamples*upsampleFactor*sizeof(float));

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
      hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  return d_U;
}

void cleanupDeviceMemory(float * d_U)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Free device global memory
    err = hipFree(d_U);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector (error code %s)!\n",
        hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void deviceUpsample(float *output, float const *input, int numElements, int upsampleFactor)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        int startingIndex = i*upsampleFactor;
        output[startingIndex] = input[i];

        for(int i = 1; i < upsampleFactor; i++)
        {
            output[startingIndex+i] = 0;
        }
   }
}

void execute(float *d_Output, float const *d_Input, int numElements, int upsampleFactor)
{
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // Create CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start, 0); // 0 indicates the default stream

    deviceUpsample<<<blocksPerGrid, threadsPerBlock>>>(d_Output, d_Input, numElements, upsampleFactor);

    // Record the stop event
    hipEventRecord(stop, 0);

    // Synchronize the stop event to ensure all preceding operations in the stream are complete
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    //printf("upSample Kernel execution time: %f ms\n", milliseconds);

    // Destroy the events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

} //namespace upSample 